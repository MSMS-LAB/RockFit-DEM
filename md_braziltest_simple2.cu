#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <math_functions.h>
#include "md.h"
#include "pcuda_helper.h"
#include "md_math_constants.h"
#include "md_phys_constants.h"
#include "md_definedparams.h"
#include <thrust/count.h>
#include <thrust/device_allocator.h>
#include <thrust/device_ptr.h>
#include <time.h>

//#include <cudpp.h>
//#include <cudpp_plan.h>

__global__ void d_braziltest_simple2(float* __restrict__ R, float* V, float* F, float* __restrict__ FL, const uint_fast32_t N, const float3 c, const float RR, const float Yt, const float Yb, const float Ytr, const float Ybr, const float vt, const float Zcut)
{
	__shared__ float s_mem[2 * SMEMDIM];
	uint_fast32_t idx = blockIdx.x * blockDim.x + threadIdx.x, tid = threadIdx.x;	
	//uint3 cm;
	//if (idx == 0)printf("In %e %e %e %e |%e %e %e %e | %e %e %e %e | %f %f\n", FV[idx], FV[idx + n], FU[idx], FU[idx + n], VV[idx], VV[idx + n], VU[idx], VU[idx + n], V[idx], V[idx + n], U[idx], U[idx + n], P_dtm, P_dt);
	//if(blockIdx.x > 3)printf("Inc %u %u %u %u %u\n", idx, n, threadIdx.x, blockIdx.x, blockDim.x);
	float3 r;
	float sfb = 0, sft = 0, _1d_v;
	while (idx < N)
	{
		r.x = R[idx] - c.x;
		r.y = R[idx + N];
		r.z = R[idx + 2 * N] - c.z;
		//printf("I0n %u %e | %e | %e %e %e\n", idx, A.z, B.z, hp.x, hp.y, hp.z);
		if (r.x * r.x + r.z * r.z < RR && r.z < Zcut)
		{
			if (r.y - Ytr > -1e-9)
			{				
				
				float fy = F[idx + N];
				if (fy > 0)
				{
					sft += fy;
					_1d_v = V[idx] * V[idx] + V[idx + 2 * N] * V[idx + 2 * N];
					if (_1d_v > 1e-9)
					{
						_1d_v = __frsqrt_rn(_1d_v);
						F[idx] -= 0.45f * fy * V[idx] * _1d_v;
						F[idx + 2 * N] -= 0.45f * fy * V[idx + 2 * N] * _1d_v;
					}
					
						
				}
				if (r.y - Yt > -1e-9)
				{
					R[idx + N] = Yt;
					//V[idx] = 0;
					//V[idx + N] = 0;
					V[idx + N] = vt;					
					F[idx + N] = 0;
				}
				//printf("Uft %u %e %e %e\n", idx, (Hd2 - r.z) * C, fz, R[idx + 2 * N]);
			}
				
			if (r.y - Ybr < 1e-9)
			{
				float fy = F[idx + N];
				if (fy < 0)
				{
					sfb += fy;
					_1d_v = V[idx] * V[idx] + V[idx + 2 * N] * V[idx + 2 * N];
					if (_1d_v > 1e-9)
					{
						_1d_v = __frsqrt_rn(_1d_v);
						F[idx] += 0.45f * fy * V[idx] * _1d_v;
						F[idx + 2 * N] += 0.45f * fy * V[idx + 2 * N] * _1d_v;
					}
				}			
				if (r.y - Yb < 1e-9)
				{					
					R[idx + N] = Yb;
					//V[idx] = 0;
					//V[idx + N] = 0;
					V[idx + N] = 0;					
					F[idx + N] = 0;
					//printf("Ufb %u %e %e | %e %e %e %e\n", idx, (Hd2 + r.z) * C, fz, R[idx + 2 * N], c.z, r.z, Hd2);
				}
				//printf("Ufb %u %e %e | %e %e %e %e\n", idx, (Hd2 + r.z) * C, fz, R[idx + 2 * N], c.z, r.z, Hd2);
			}				
		}
		//printf("I0n %u %e %e %e\n", idx, R[idx], R[idx + N], R[idx + 2 * N]);
		idx += blockDim.x * gridDim.x;
	}
	s_mem[tid] = sfb;
	s_mem[tid + SMEMDIM] = sft;
	__syncthreads();

	if (blockDim.x >= 1024 && tid < 512)
	{
		s_mem[tid] += s_mem[tid + 512];
		s_mem[tid + SMEMDIM] += s_mem[tid + SMEMDIM + 512];		
	}
	__syncthreads();

	if (blockDim.x >= 512 && tid < 256)
	{
		//printf("Blok!\n");
		s_mem[tid] += s_mem[tid + 256];
		s_mem[tid + SMEMDIM] += s_mem[tid + SMEMDIM + 256];		
	}
	__syncthreads();

	if (blockDim.x >= 256 && tid < 128)
	{
		s_mem[tid] += s_mem[tid + 128];
		s_mem[tid + SMEMDIM] += s_mem[tid + SMEMDIM + 128];		
	}
	__syncthreads();

	if (blockDim.x >= 128 && tid < 64)
	{
		s_mem[tid] += s_mem[tid + 64];
		s_mem[tid + SMEMDIM] += s_mem[tid + SMEMDIM + 64];		
	}
	__syncthreads();
	
	// unrolling warp
	if (tid < 32)
	{
		volatile float* vsmem = s_mem;
		vsmem[tid] += vsmem[tid + 32];
		vsmem[tid + SMEMDIM] += vsmem[tid + SMEMDIM + 32];		
		vsmem[tid] += vsmem[tid + 16];
		vsmem[tid + SMEMDIM] += vsmem[tid + SMEMDIM + 16];		
		vsmem[tid] += vsmem[tid + 8];
		vsmem[tid + SMEMDIM] += vsmem[tid + SMEMDIM + 8];		
		vsmem[tid] += vsmem[tid + 4];
		vsmem[tid + SMEMDIM] += vsmem[tid + SMEMDIM + 4];		
		vsmem[tid] += vsmem[tid + 2];
		vsmem[tid + SMEMDIM] += vsmem[tid + SMEMDIM + 2];		
		vsmem[tid] += vsmem[tid + 1];
		vsmem[tid + SMEMDIM] += vsmem[tid + SMEMDIM + 1];		
	}

	// write result for this block to global mem
	if (tid == 0)
	{
		//printf("U %u %u %u\n", idx, blockIdx.x, blockIdx.x + gridDim.x);
		FL[blockIdx.x] = s_mem[0];
		FL[blockIdx.x + gridDim.x] = s_mem[SMEMDIM];		
	}
}
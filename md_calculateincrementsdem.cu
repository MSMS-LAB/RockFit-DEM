#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <math_functions.h>
#include "md.h"
#include "pcuda_helper.h"
#include "md_math_constants.h"
#include "md_phys_constants.h"
#include <thrust/count.h>
#include <thrust/device_allocator.h>
#include <thrust/device_ptr.h>
#include <time.h>

//#include <cudpp.h>
//#include <cudpp_plan.h>


__global__ void d_CalculateIncrementsDEM(const float* __restrict__ F, float* __restrict__ V, float* __restrict__ R,
	const float* __restrict__ M, float* __restrict__ W,
	const uint_fast32_t N, const float _1d_Mass_m_dt, const float dt, const float _1d_I_m_dt)
{
	uint_fast32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	//if (idx == 0)printf("In %e %e %e %e |%e %e %e %e | %e %e %e %e | %f %f\n", FV[idx], FV[idx + n], FU[idx], FU[idx + n], VV[idx], VV[idx + n], VU[idx], VU[idx + n], V[idx], V[idx + n], U[idx], U[idx + n], P_dtm, P_dt);
	//if(blockIdx.x > 3)printf("Inc %u %u %u %u %u\n", idx, n, threadIdx.x, blockIdx.x, blockDim.x);
	while (idx < N)
	{	
		//printf("In %u | %e %e %e | %e %e %e \n", idx, F[idx], F[idx + N], F[idx + 2 * N], M[idx], M[idx + N], M[idx + 2 * N]);
		//printf("InF %u | %e %e %e | %e %e %e \n", idx, V[idx], V[idx + N], V[idx + 2 * N], F[idx], F[idx + N], F[idx + 2 * N]);

		V[idx] += __fmul_rn(F[idx], _1d_Mass_m_dt);
		V[idx + N] += __fmul_rn(F[idx + N], _1d_Mass_m_dt);
		V[idx + 2 * N] += __fmul_rn(F[idx + 2 * N], _1d_Mass_m_dt);

		R[idx] += __fmul_rn(V[idx], dt);
		R[idx + N] += __fmul_rn(V[idx + N], dt);
		R[idx + 2 * N] += __fmul_rn(V[idx + 2 * N], dt);

		W[idx] += __fmul_rn(M[idx], _1d_I_m_dt);
		W[idx + N] += __fmul_rn(M[idx + N], _1d_I_m_dt);
		W[idx + 2 * N] += __fmul_rn(M[idx + 2 * N], _1d_I_m_dt);
		//W[idx] = 0;
		//W[idx + N] = 0;
		//W[idx + 2 * N] = 0;
		//printf("InU %u | %e %e %e | %e %e %e \n", idx, __fmul_rn(V[idx], dt), __fmul_rn(V[idx + N], dt), __fmul_rn(V[idx + 2 * N], dt), __fmul_rn(M[idx], _1d_I_m_dt), __fmul_rn(M[idx + N], _1d_I_m_dt), __fmul_rn(M[idx + 2 * N], _1d_I_m_dt));

		idx += blockDim.x * gridDim.x;
	}	
}